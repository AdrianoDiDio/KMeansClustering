#include "hip/hip_runtime.h"
/*
    PlaceHolder
*/
extern "C" {
#include "KMeansClustering.h"
}

#define CUDA_CHECK_RETURN(Value) CheckCudaErrorAux(__FILE__,__LINE__, #Value, Value)
void CheckCudaErrorAux(const char *File, unsigned Line,const char *Statement, hipError_t ErrorCode)
{
   if (ErrorCode != hipSuccess) 
   {
      fprintf(stderr,"CheckCudaErrorAux: %s returned %s %s %d\n",Statement,hipGetErrorString(ErrorCode), File, Line);
      exit(ErrorCode);
   }
}

__global__ void CompareCentroidsKernel(int *Sum,float *Centroids,float *OldCentroids,int NumCentroids,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    float Delta;
    int Value;
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;

    if( ThreadIndexX < NumCentroids ) {
        Delta = fabsf(Centroids[ThreadIndexX * Stride + ThreadIndexY] - OldCentroids[ThreadIndexX * Stride + ThreadIndexY]);
        Value = Delta < KMEANS_ALGORITHM_TOLERANCE ? 1 : 0;
        atomicAdd(Sum,Value);
    }
}
__global__ void MeanPointsInClustersKernel(float *Centroids,int NumCentroids,float *ClusterCounter,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    int NumClusters;
    
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    NumClusters = ClusterCounter[ThreadIndexX];
    if( ThreadIndexX < NumCentroids && NumClusters != 0) {
        Centroids[ThreadIndexX * Stride + ThreadIndexY] /= ( ClusterCounter[ThreadIndexX] / Stride );
    }
}
__global__ void SumPointsInClustersKernel(float *Centroids,int NumCentroids,float *ClusterCounter,
                                          int *Clusters,float *Points,int NumPoints,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    int CentroidIndex;
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    if( ThreadIndexX < NumPoints && ThreadIndexY < Stride ) {
        CentroidIndex = Clusters[ThreadIndexX];
        atomicAdd(&(Centroids[CentroidIndex * Stride + ThreadIndexY]),Points[ThreadIndexX * Stride + ThreadIndexY]);
        atomicAdd(&(ClusterCounter[CentroidIndex]), 1.f);
    }
}
__global__ void BuildClusterListKernel(int *Clusters,float *Distances,int NumPoints,int NumCentroids,int Stride)
{
    int ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    int DistanceArrayBaseIndex = ThreadIndexX * NumCentroids;
    float Min;
    int MinIndex;
    int i;

    if( ThreadIndexX < NumPoints ) {
        Min = INFINITY;
        MinIndex = 0;
        for( i = 0; i < NumCentroids; i++ ) {
            if( Distances[DistanceArrayBaseIndex + i] < Min ) {
                Min = Distances[DistanceArrayBaseIndex + i];
                MinIndex = i;
            }
        }
        Clusters[ThreadIndexX] = MinIndex;
    }
}

__global__ void ClusterComputeDistanceSquaredKernel(float *Distances,
    float *Centroids,int NumCentroids,float *Points,int NumPoints,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    int CentroidIndex;
    int DatasetIndex;
    int i;
    float LocalDistance;
    
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    if( ThreadIndexX < NumPoints && ThreadIndexY < NumCentroids ) {
    LocalDistance = 0.f;
        for( i = 0; i < Stride; i++ ) {
            CentroidIndex = ThreadIndexY * Stride + i;
            DatasetIndex = ThreadIndexX  * Stride + i;
            LocalDistance += (Centroids[CentroidIndex] - Points[DatasetIndex]) * 
                        (Centroids[CentroidIndex] - Points[DatasetIndex]);
        }
        Distances[ThreadIndexX * NumCentroids + ThreadIndexY] = LocalDistance;
    }
}

__global__ void CentroidsInitKernel(float *Centroids,int NumCentroids, float *Points,int NumPoints,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    if( ThreadIndexX < NumCentroids && ThreadIndexY < Stride) {
        Centroids[ThreadIndexX * Stride + ThreadIndexY] = Points[ThreadIndexX * Stride + ThreadIndexY];
    }
}

int CudaCompareCentroidsList(int *DeviceSum,float *DeviceCentroidList,float *DeviceOldCentroidList,int NumCentroids,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;
    int    Sum;

    BlockSize = dim3(256,Stride,1);
    GridSize = dim3((NumCentroids + BlockSize.x - 1) / BlockSize.x,1,1);
    
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceSum,0,sizeof(int)));
    CompareCentroidsKernel<<<GridSize,BlockSize>>>(DeviceSum,DeviceCentroidList,DeviceOldCentroidList,NumCentroids,Stride);

    CUDA_CHECK_RETURN(hipMemcpy(&Sum,DeviceSum,sizeof(int),hipMemcpyDeviceToHost));
    
    return Sum == (NumCentroids * Stride) ? 1 : 0;
}
void CudaComputeDistances(float *DeviceDistanceList,float *DeviceCentroidList,int NumCentroids,float *DevicePointList,
                            int NumPoints,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;
    BlockSize = dim3(32,32,1);
    GridSize = dim3((NumPoints + BlockSize.x - 1) / BlockSize.x ,(NumCentroids + BlockSize.y - 1) / BlockSize.y ,1);
    ClusterComputeDistanceSquaredKernel<<<GridSize,BlockSize>>>
        (DeviceDistanceList,DeviceCentroidList,NumCentroids,DevicePointList,NumPoints,Stride);
}
void CudaBuildClusterList(int *DeviceClusterList,float *DeviceDistanceList,int NumPoints,int NumCentroids,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;

    BlockSize = dim3(256,1,1);
    GridSize = dim3((NumPoints + BlockSize.x - 1) / BlockSize.x,1,1);
    BuildClusterListKernel<<<GridSize,BlockSize>>>(DeviceClusterList,DeviceDistanceList,NumPoints,NumCentroids,Stride);
}
void CudaSumPointsInClusters(float *DeviceCentroidList,int NumCentroids,int *DeviceClusterList,float *DeviceClusterCounter,
                             float *DevicePointList,int NumPoints,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;

    BlockSize = dim3(256,Stride,1);
    GridSize = dim3((NumPoints + BlockSize.x - 1) / BlockSize.x,1,1);

    SumPointsInClustersKernel<<<GridSize,BlockSize>>>(DeviceCentroidList,NumCentroids,DeviceClusterCounter,
                                                      DeviceClusterList,DevicePointList,NumPoints,Stride);
}

void CudaMeanPointsInClusters(float *DeviceCentroidList,int NumCentroids,float *DeviceClusterCounter,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;
    
    BlockSize = dim3(256,Stride,1);
    GridSize = dim3((NumCentroids + BlockSize.x - 1) / BlockSize.x,1,1);
    
    MeanPointsInClustersKernel<<<GridSize,BlockSize>>>(DeviceCentroidList,NumCentroids,DeviceClusterCounter,Stride);
}


void CudaUpdateCentroidList(float *DeviceCentroidList,int NumCentroids,float *DevicePoints,int NumPoints,int *DeviceClusterList,
                            float *DeviceClusterCounter,int Stride)
{
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceCentroidList,0,NumCentroids * Stride * sizeof(float)));
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceClusterCounter,0,NumCentroids * sizeof(float)));
    
    CudaSumPointsInClusters(DeviceCentroidList,NumCentroids,DeviceClusterList,DeviceClusterCounter,DevicePoints,
                            NumPoints,Stride);
    CudaMeanPointsInClusters(DeviceCentroidList,NumCentroids,DeviceClusterCounter,Stride);
}
float *CudaInitCentroids(int NumCentroids,float *DevicePointList,int NumPoints,int Stride)
{
    float *DeviceCentroidOutputList;
    int    CentroidListSize;
    dim3   BlockSize;
    dim3   GridSize;

    CentroidListSize = NumCentroids * Stride * sizeof(float);
    
    CUDA_CHECK_RETURN(hipMalloc((void**)&DeviceCentroidOutputList,CentroidListSize));
    
    BlockSize = dim3(64, Stride, 1);
    GridSize = dim3((NumCentroids + BlockSize.x - 1) / BlockSize.x,1,1);
    
    CentroidsInitKernel<<<GridSize,BlockSize>>>
            (DeviceCentroidOutputList,NumCentroids,DevicePointList,
             NumPoints,Stride);
    
    return DeviceCentroidOutputList;
}

float *CudaInitClusterCounter(int NumCentroids)
{
    float *DeviceClusterCounter;
    int  ClusterCounterSize;

    ClusterCounterSize = NumCentroids * sizeof(float);

    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceClusterCounter, ClusterCounterSize));
    
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceClusterCounter,0,ClusterCounterSize));

    return DeviceClusterCounter; 
}
int *CudaInitAssignments(PointArrayList_t *PointList)
{
    int *DeviceClusterList;
    int    ClusterListSize;

    ClusterListSize = PointList->NumPoints * sizeof(int);

    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceClusterList, ClusterListSize));
    
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceClusterList,-1,ClusterListSize));

    return DeviceClusterList;
}
float *CudaInitDistances(int NumCentroids,int NumPoints)
{
    float *DeviceDistanceList;
    int    DistanceListSize;

    DistanceListSize = NumPoints * NumCentroids * sizeof(float);

    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceDistanceList, DistanceListSize));
    
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceDistanceList,0.f,DistanceListSize));

    return DeviceDistanceList;
}
float *CudaInitPointList(PointArrayList_t *PointList)
{
    float *DevicePointList;
    int    PointListSize;
    
    PointListSize = PointList->NumPoints * PointList->Stride * sizeof(float);
    CUDA_CHECK_RETURN(hipMalloc((void **)&DevicePointList, PointListSize));
    CUDA_CHECK_RETURN(
        hipMemcpy(DevicePointList,PointList->Points, 
                   PointListSize,hipMemcpyHostToDevice));
    return DevicePointList;
}
void CudaMain(int NumCentroids,int Stride,PointArrayList_t *PointList)
{
    float *DevicePointList;
    float *DeviceCentroidList;
    float *DeviceOldCentroidList;
    float *DeviceDistanceList;
    int   *DeviceClusterList;
    int    OldCentroidListSize;
    float *DeviceClusterCounter;
    int   *DeviceSum;
    int    Sum;
    int    SumSize;
    int    Step;
    
    DevicePointList = CudaInitPointList(PointList);
    DeviceCentroidList = CudaInitCentroids(NumCentroids,DevicePointList,PointList->NumPoints,Stride);
    OldCentroidListSize = NumCentroids * Stride * sizeof(float);
    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceOldCentroidList,OldCentroidListSize));
    DeviceClusterCounter = CudaInitClusterCounter(NumCentroids);
    DeviceClusterList = CudaInitAssignments(PointList);
    DeviceDistanceList = CudaInitDistances(NumCentroids,PointList->NumPoints);
    SumSize = sizeof(int);
    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceSum,SumSize));
    
    Step = 0;
    
    while( 1 ) {
        CudaComputeDistances(DeviceDistanceList,DeviceCentroidList,NumCentroids,DevicePointList,PointList->NumPoints,Stride);
        CudaBuildClusterList(DeviceClusterList,DeviceDistanceList,PointList->NumPoints,NumCentroids,Stride);
        hipMemcpy(DeviceOldCentroidList,DeviceCentroidList,OldCentroidListSize,hipMemcpyDeviceToDevice);
        CudaUpdateCentroidList(DeviceCentroidList,NumCentroids,DevicePointList,PointList->NumPoints,DeviceClusterList,
                            DeviceClusterCounter,Stride);
        Sum = CudaCompareCentroidsList(DeviceSum,DeviceCentroidList,DeviceOldCentroidList,NumCentroids,Stride);
        if( Sum == 1 ) {
            break;
        }
        Step++;
    }

    hipFree(DevicePointList);
    hipFree(DeviceCentroidList);
    hipFree(DeviceOldCentroidList);
    hipFree(DeviceDistanceList);
    hipFree(DeviceClusterList);
    hipFree(DeviceClusterCounter);
}
int main(int argc,char** argv)
{
    PointArrayList_t *PointList;
    long Start;
    long Delta;
    int Stride;
    PointList = LoadPointsDataset(&Stride);
    
    if( PointList == NULL ) {
        DPrintf("Couldn't load point dataset.\n");
        return -1;
    }
    Start = SysMilliseconds();
    CudaMain(1000,Stride,PointList);
    Delta = SysMilliseconds() - Start;
	printf("Time: %f seconds\r\n", Delta * 0.001f);
    
    
    PointArrayListCleanUp(PointList);
    
    free(PointList);
}

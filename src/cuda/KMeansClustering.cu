#include "hip/hip_runtime.h"
/*
    PlaceHolder
*/
extern "C" {
#include "KMeansClustering.h"
}

#define CUDA_CHECK_RETURN(Value) CheckCudaErrorAux(__FILE__,__LINE__, #Value, Value)
void CheckCudaErrorAux(const char *File, unsigned Line,const char *Statement, hipError_t ErrorCode)
{
   if (ErrorCode != hipSuccess) 
   {
      fprintf(stderr,"CheckCudaErrorAux: %s returned %s %s %d\n",Statement,hipGetErrorString(ErrorCode), File, Line);
      exit(ErrorCode);
   }
}

__global__ void CompareCentroidsKernel(int *Sum,float *Centroids,float *OldCentroids,int NumCentroids,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    float Delta;
    int Value;
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;

    if( ThreadIndexX < NumCentroids ) {
        Delta = fabsf(Centroids[ThreadIndexX * Stride + ThreadIndexY] - OldCentroids[ThreadIndexX * Stride + ThreadIndexY]);
        Value = Delta < KMEANS_ALGORITHM_TOLERANCE ? 1 : 0;
        atomicAdd(Sum,Value);
    }
}
__global__ void MeanPointsInClustersKernel(float *Centroids,int NumCentroids,float *ClusterCounter,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    int NumClusters;
    
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    NumClusters = ClusterCounter[ThreadIndexX];
    if( ThreadIndexX < NumCentroids && NumClusters != 0) {
        Centroids[ThreadIndexX * Stride + ThreadIndexY] /= ( ClusterCounter[ThreadIndexX] / Stride );
    }

}
__global__ void SumPointsInClustersKernel(float *Centroids,int NumCentroids,float *ClusterCounter,
                                          int *Clusters,float *Points,int NumPoints,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    int CentroidIndex;
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    if( ThreadIndexX < NumPoints && ThreadIndexY < Stride ) {
        CentroidIndex = Clusters[ThreadIndexX];
        atomicAdd(&(Centroids[CentroidIndex * Stride + ThreadIndexY]),Points[ThreadIndexX * Stride + ThreadIndexY]);
        //NOTE(Adriano):This gets added Stride times for each point...
        atomicAdd(&(ClusterCounter[CentroidIndex]), 1.f);
    }
}
__global__ void BuildClusterListKernel(int *Clusters,float *Distances,int NumPoints,int NumCentroids,int Stride)
{
    int ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    int DistanceArrayBaseIndex = ThreadIndexX * NumCentroids;
    float Min;
    int MinIndex;
    int i;

    
    if( ThreadIndexX < NumPoints ) {
        Min = INFINITY;
        MinIndex = 0;
        for( i = 0; i < NumCentroids; i++ ) {
            if( Distances[DistanceArrayBaseIndex + i] < Min ) {
                Min = Distances[DistanceArrayBaseIndex + i];
                MinIndex = i;
            }
        }
        Clusters[ThreadIndexX] = MinIndex;
    }
}

__global__ void ClusterComputeDistanceSquaredKernel(float *Distances,
    float *Centroids,int NumCentroids,float *Points,int NumPoints,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    int CentroidIndex;
    int DatasetIndex;
    int i;
    float LocalDistance;
    
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    if( ThreadIndexX < NumPoints && ThreadIndexY < NumCentroids ) {
        LocalDistance = 0.f;
        for( i = 0; i < Stride; i++ ) {
            CentroidIndex = ThreadIndexY * Stride + i;
            DatasetIndex = ThreadIndexX  * Stride + i;
            LocalDistance += (Centroids[CentroidIndex] - Points[DatasetIndex]) * 
                        (Centroids[CentroidIndex] - Points[DatasetIndex]);
        }
        //NOTE(Adriano):Store it flat, in an array which is made of blocks
        //of size equal to the number of points in the dataset
        //where each block contains a float distance for each centroid.
        Distances[ThreadIndexX * NumCentroids + ThreadIndexY] = LocalDistance;
    }
}

__global__ void CentroidsInitKernel(float *Centroids,int NumCentroids, float *Points,int NumPoints,int Stride)
{
    int ThreadIndexX;
    int ThreadIndexY;
    ThreadIndexX = blockIdx.x * blockDim.x + threadIdx.x;
    ThreadIndexY = blockIdx.y * blockDim.y + threadIdx.y;
    if( ThreadIndexX < NumCentroids && ThreadIndexY < Stride) {
        //Assign it
        Centroids[ThreadIndexX * Stride + ThreadIndexY] = Points[ThreadIndexX * Stride + ThreadIndexY];
    }
    //Centroids[] => Points[]
}
/*
    Check if we have converged by comparing the old centroid array with the
    updated one.
    Returns 1 if the algorithm has converged (Centroid's position are not changing anymore) or
    0 if not.
    The way it does it's by subtracting the two arrays elements and adding
    a 1 if the difference is greater than the defined threshold or 0 if not.
    When the kernel has finished, if sum is equal to (NumCentroids*Stride) then
    we know that all centroid's position have not changed much.
*/
int CudaCompareCentroidsList(int *DeviceSum,float *DeviceCentroidList,float *DeviceOldCentroidList,int NumCentroids,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;
    int    Sum;

    BlockSize = dim3(256,Stride,1);
    GridSize = dim3((NumCentroids + BlockSize.x - 1) / BlockSize.x,1,1);
    
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceSum,0,sizeof(int)));
    CompareCentroidsKernel<<<GridSize,BlockSize>>>(DeviceSum,DeviceCentroidList,DeviceOldCentroidList,NumCentroids,Stride);

//     CUDA_CHECK_RETURN( hipPeekAtLastError() );
//     CUDA_CHECK_RETURN( hipDeviceSynchronize() );
    
    CUDA_CHECK_RETURN(hipMemcpy(&Sum,DeviceSum,sizeof(int),hipMemcpyDeviceToHost));
    
    return Sum == (NumCentroids * Stride) ? 1 : 0;
}
void CudaComputeDistances(float *DeviceDistanceList,float *DeviceCentroidList,int NumCentroids,float *DevicePointList,
                            int NumPoints,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;

    BlockSize = dim3(32,32,1);
    GridSize = dim3((NumPoints + BlockSize.x - 1) / BlockSize.x ,(NumCentroids + BlockSize.y - 1) / BlockSize.y ,1);
    ClusterComputeDistanceSquaredKernel<<<GridSize,BlockSize>>>
        (DeviceDistanceList,DeviceCentroidList,NumCentroids,DevicePointList,NumPoints,Stride);

//     CUDA_CHECK_RETURN( hipPeekAtLastError() );
//     CUDA_CHECK_RETURN( hipDeviceSynchronize() );
//     return DeviceCentroidList;
}
void CudaBuildClusterList(int *DeviceClusterList,float *DeviceDistanceList,int NumPoints,int NumCentroids,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;

    BlockSize = dim3(256,1,1);
    GridSize = dim3((NumPoints + BlockSize.x - 1) / BlockSize.x,1,1);
    BuildClusterListKernel<<<GridSize,BlockSize>>>(DeviceClusterList,DeviceDistanceList,NumPoints,NumCentroids,Stride);

//     CUDA_CHECK_RETURN( hipPeekAtLastError() );
//     CUDA_CHECK_RETURN( hipDeviceSynchronize() );
//     return DeviceClusterList;
}
void CudaSumPointsInClusters(float *DeviceCentroidList,int NumCentroids,int *DeviceClusterList,float *DeviceClusterCounter,
                             float *DevicePointList,int NumPoints,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;

    BlockSize = dim3(256,Stride,1);
    GridSize = dim3((NumPoints + BlockSize.x - 1) / BlockSize.x,1,1);

    SumPointsInClustersKernel<<<GridSize,BlockSize>>>(DeviceCentroidList,NumCentroids,DeviceClusterCounter,
                                                      DeviceClusterList,DevicePointList,NumPoints,Stride);

//     CUDA_CHECK_RETURN( hipPeekAtLastError() );
//     CUDA_CHECK_RETURN( hipDeviceSynchronize() );
}

void CudaMeanPointsInClusters(float *DeviceCentroidList,int NumCentroids,float *DeviceClusterCounter,int Stride)
{
    dim3   BlockSize;
    dim3   GridSize;

    BlockSize = dim3(256,Stride,1);
    GridSize = dim3((NumCentroids + BlockSize.x - 1) / BlockSize.x,1,1);

    MeanPointsInClustersKernel<<<GridSize,BlockSize>>>(DeviceCentroidList,NumCentroids,DeviceClusterCounter,Stride);

//     CUDA_CHECK_RETURN( hipPeekAtLastError() );
//     CUDA_CHECK_RETURN( hipDeviceSynchronize() );
}


void CudaUpdateCentroidList(float *DeviceCentroidList,int NumCentroids,float *DevicePoints,int NumPoints,int *DeviceClusterList,
                            float *DeviceClusterCounter,int Stride)
{
    
    //Make sure we zero out the centroid's position...
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceCentroidList,0,NumCentroids * Stride * sizeof(float)));
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceClusterCounter,0,NumCentroids * sizeof(float)));

    CudaSumPointsInClusters(DeviceCentroidList,NumCentroids,DeviceClusterList,DeviceClusterCounter,DevicePoints,
                            NumPoints,Stride);
    CudaMeanPointsInClusters(DeviceCentroidList,NumCentroids,DeviceClusterCounter,Stride);
    //CudaGet
}
float *CudaInitCentroids(int NumCentroids,float *DevicePointList,int NumPoints,int Stride)
{
    float *DeviceCentroidOutputList;
    int    CentroidListSize;
    dim3   BlockSize;
    dim3   GridSize;

    //Step.1 Initialize all the centroids.
    CentroidListSize = NumCentroids * Stride * sizeof(float);


    CUDA_CHECK_RETURN(hipMalloc((void**)&DeviceCentroidOutputList,CentroidListSize));

    BlockSize = dim3(64, Stride, 1);
    GridSize = dim3((NumCentroids + BlockSize.x - 1) / BlockSize.x,1,1);
    CentroidsInitKernel<<<GridSize,BlockSize>>>(DeviceCentroidOutputList,NumCentroids,DevicePointList,NumPoints,Stride);

//     CUDA_CHECK_RETURN( hipPeekAtLastError() );
//     CUDA_CHECK_RETURN( hipDeviceSynchronize() );
    return DeviceCentroidOutputList;
}

float *CudaInitClusterCounter(int NumCentroids)
{
    float *DeviceClusterCounter;
    int  ClusterCounterSize;

    ClusterCounterSize = NumCentroids * sizeof(float);

    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceClusterCounter, ClusterCounterSize));
    
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceClusterCounter,0,ClusterCounterSize));

    return DeviceClusterCounter; 
}
int *CudaInitAssignments(PointArrayList_t *PointList)
{
    int *DeviceClusterList;
    int    ClusterListSize;

    ClusterListSize = PointList->NumPoints * sizeof(int);

    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceClusterList, ClusterListSize));
    
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceClusterList,-1,ClusterListSize));

    return DeviceClusterList;
}
float *CudaInitDistances(int NumCentroids,int NumPoints)
{
    float *DeviceDistanceList;
    int    DistanceListSize;

    DistanceListSize = NumPoints * NumCentroids * sizeof(float);

    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceDistanceList, DistanceListSize));
    
    CUDA_CHECK_RETURN(hipMemset((void *)DeviceDistanceList,0.f,DistanceListSize));

    return DeviceDistanceList;
}
float *CudaInitPointList(PointArrayList_t *PointList)
{
    float *DevicePointList;
    int    PointListSize;

    PointListSize = PointList->NumPoints * PointList->Stride * sizeof(float);
    CUDA_CHECK_RETURN(hipMalloc((void **)&DevicePointList, PointListSize));
    CUDA_CHECK_RETURN(hipMemcpy(DevicePointList,PointList->Points, PointListSize,
                                 hipMemcpyHostToDevice));
    return DevicePointList;
}
void CudaMain(int NumCentroids,int Stride,PointArrayList_t *PointList)
{
    float *DevicePointList;
    float *DeviceCentroidList;
    float *DeviceOldCentroidList;
    float *DeviceDistanceList;
    int   *DeviceClusterList;
    int    OldCentroidListSize;
    float *DeviceClusterCounter;
    int   *DeviceSum;
    int    Sum;
    int    SumSize;
    //DEBUG
    float *DebugOutputList;
    int    DebugOutputListSize;
    int   *DebugOutputList2;
    int    DebugOutputListSize2;
    float  *DebugOutputList3;
    int    DebugOutputListSize3;
    float  *DebugOutputList4;
    int    DebugOutputListSize4;
    int Step;
    int Start;
    int End;
//     hipProfilerStart();
    DebugOutputListSize = PointList->NumPoints * NumCentroids * sizeof(float);
    DebugOutputList = (float *) malloc(DebugOutputListSize);
    
    DebugOutputListSize2 = PointList->NumPoints * sizeof(int);
    DebugOutputList2 = (int *) malloc(DebugOutputListSize2);
    
    DebugOutputListSize3 = PointList->NumPoints * Stride * sizeof(float);
    DebugOutputList3 = (float *) malloc(DebugOutputListSize3);
    
    DebugOutputListSize4 = NumCentroids * Stride * sizeof(float);
    DebugOutputList4 = (float *) malloc(DebugOutputListSize4);

    
    assert(PointList->Stride == Stride);
    
    Start = SysMilliseconds();
    
    DevicePointList = CudaInitPointList(PointList);
    DeviceCentroidList = CudaInitCentroids(NumCentroids,DevicePointList,PointList->NumPoints,Stride);
    
    OldCentroidListSize = NumCentroids * Stride * sizeof(float);
    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceOldCentroidList,OldCentroidListSize));
    
    DeviceClusterCounter = CudaInitClusterCounter(NumCentroids);
    
    DeviceClusterList = CudaInitAssignments(PointList);
    DeviceDistanceList = CudaInitDistances(NumCentroids,PointList->NumPoints);
    SumSize = sizeof(int);
    CUDA_CHECK_RETURN(hipMalloc((void **)&DeviceSum,SumSize));
    Step = 0;
    while( 1 ) {
        CudaComputeDistances(DeviceDistanceList,DeviceCentroidList,NumCentroids,DevicePointList,PointList->NumPoints,Stride);
        CudaBuildClusterList(DeviceClusterList,DeviceDistanceList,PointList->NumPoints,NumCentroids,Stride);

    #if 0/* _DEBUG*/
        //Test to check if the Distance and BuildClusterList Kernel are working as intended.
        hipMemcpy(DebugOutputList, DeviceDistanceList, DebugOutputListSize,hipMemcpyDeviceToHost);
        hipMemcpy(DebugOutputList2, DeviceClusterList, DebugOutputListSize2,hipMemcpyDeviceToHost);
        int PointNumber = 0;
        float Min;
        int SelectedCentroid;
        for( int i = 0; i < PointList->NumPoints * NumCentroids; i+=NumCentroids ) {
            DPrintf("Point %i Distances: \n",PointNumber);
            Min = INFINITY;
            SelectedCentroid = -1;
            for( int j = 0; j < NumCentroids; j++ ) {
                DPrintf("Centroid (i == %i) %i:%f\n",i,j,DebugOutputList[i+j]);
                if( DebugOutputList[i+j] < Min ) {
                    Min = DebugOutputList[i+j];
                    SelectedCentroid = j;
                }
            }
    //         DPrintf("Point %i has chosen centroid %i with a min distance of %f\n",PointNumber,SelectedCentroid,Min);
    //         DPrintf("In Cuda List was %i\n",DebugOutputList2[i]);
    //         if( SelectedCentroid != DebugOutputList2[PointNumber] ) {
    //             DPrintf("Mismatched Cuda/Linear Centroid selection...Expected %i found %i\n",
    //                 SelectedCentroid,DebugOutputList2[PointNumber]);
    //         }
            assert( SelectedCentroid == DebugOutputList2[PointNumber] );
            PointNumber++;
        }
    #endif
        hipMemcpy(DeviceOldCentroidList,DeviceCentroidList,OldCentroidListSize,hipMemcpyDeviceToDevice);
        CudaUpdateCentroidList(DeviceCentroidList,NumCentroids,DevicePointList,PointList->NumPoints,DeviceClusterList,
                            DeviceClusterCounter,Stride);
        Sum = CudaCompareCentroidsList(DeviceSum,DeviceCentroidList,DeviceOldCentroidList,NumCentroids,Stride);
//         hipDeviceSynchronize();

        //CentroidList
//         DPrintf("Sum is %i || Iteration:%i\n",Sum,Step);
        if( Sum == 1 ) {
            DPrintf("Reached max condition...\n");
            break;
        }
        Step++;
    }
    End = SysMilliseconds();
    printf("Cuda Completed in %i steps %i ms elapsed.\n",Step,End-Start);
    hipMemcpy(DebugOutputList4, DeviceCentroidList, DebugOutputListSize4,hipMemcpyDeviceToHost);
        //Cluster Index List
    hipMemcpy(DebugOutputList2, DeviceClusterList, DebugOutputListSize2,hipMemcpyDeviceToHost);
        //Dataset
    hipMemcpy(DebugOutputList3, DevicePointList, DebugOutputListSize3,hipMemcpyDeviceToHost);
//         hipDeviceSynchronize();
    DumpClusters(DebugOutputList3,PointList->NumPoints,DebugOutputList4,NumCentroids,DebugOutputList2,Stride,Step);

    hipFree(DevicePointList);
    hipFree(DeviceCentroidList);
    hipFree(DeviceOldCentroidList);
    hipFree(DeviceDistanceList);
    hipFree(DeviceClusterList);
    hipFree(DeviceClusterCounter);
    hipFree(DeviceSum);
    free(DebugOutputList);
    free(DebugOutputList2);
    free(DebugOutputList3);
    free(DebugOutputList4);
}
int main(int argc,char** argv)
{
    PointArrayList_t *PointList;
    int NumClusters;
    int Stride;

    if( argc != 3 ) {
        printf("Usage:%s <Dataset File> <Number of Clusters>\n",argv[0]);
        return -1;
    }
    
    PointList = LoadPointsDataset(argv[1],&Stride);
    
    if( PointList == NULL ) {
        DPrintf("Couldn't load point dataset.\n");
        return -1;
    }
    
    NumClusters = StringToInt(argv[2]);
    
    CudaMain(NumClusters,Stride,PointList);
    
    PointArrayListCleanUp(PointList);
    
    free(PointList);
}
